
#include <hip/hip_runtime.h>
#include <stdio.h>

// __global__ functions, or "kernels", execute on the device
__global__ void hello_kernel(void)
{
  printf("Hello, world from the device!\n");
}

int main(void)
{
  // greet from the host
  printf("Hello, world from the host!\n");

  // launch a kernel with a single thread to greet from the device
  hello_kernel<<<1,1>>>();

  // wait for the device to finish so that we see the message
  hipDeviceSynchronize();

  return 0;
}